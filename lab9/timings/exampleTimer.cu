#include <stdio.h>
#include <hip/hip_runtime.h>
#include <stdlib.h>
#include "helper_functions.h"


int main() {
    int *a_d
    hipMalloc((void **) &a_d, size); // alokuj pamięć na GPU
    hipMemcpy(a_d, a_h, size, hipMemcpyHostToDevice);
    StopWatchInterface *timer=NULL;
    sdkCreateTimer(&timer);
    sdkResetTimer(&timer);
    sdkStartTimer(&timer);
    // wykonaj obliczenia na GPU:
    kernel <<< n_blocks, block_size >>> (a_d, N);
    hipDeviceSynchronize();
    sdkStopTimer(&timer);
    float time = sdkGetTimerValue(&timer);
    sdkDeleteTimer(&timer);
    // prześlij wyniki
    hipMemcpy(a_h, a_d, sizeof(float)*N, hipMemcpyDeviceToHost);
    hipFree(a_d);
    printf ("Time for the kernel: %f ms\n", time);
}